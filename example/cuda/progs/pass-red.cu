
#include <hip/hip_runtime.h>
/*
 * Copyright (C) 2007 Giacomo Spigler
 * CopyPolicy: Released under the terms of the LGPLv2.1 or later, see LGPL.TXT
 */

extern "C" {

  __global__ void FragmentProgram(int w, int h, unsigned char *in, unsigned char *out) {
    int i=0;

    for(i=threadIdx.x+blockIdx.x*blockDim.x; i<w*h; i+=blockDim.x*gridDim.x) {
      if(in[i*3]>=200) {
        //out[i*3]=255;
        //out[i*3+1]=255;
        //out[i*3+2]=255;
      } else {
        out[i*3]=0;
        out[i*3+1]=0;
        out[i*3+2]=0;
      }
    }
  }

}


