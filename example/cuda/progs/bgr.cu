
#include <hip/hip_runtime.h>
/*
 * Copyright (C) 2007 Giacomo Spigler
 * CopyPolicy: Released under the terms of the LGPLv2.1 or later, see LGPL.TXT
 */

extern "C" {

  __global__ void FragmentProgram(int w, int h, unsigned char *in, unsigned char *out) {
    int i=0;

    for(i=threadIdx.x+blockIdx.x*blockDim.x; i<w*h; i+=blockDim.x*gridDim.x) {
      out[i*3]=in[i*3+2];
      out[i*3+1]=in[i*3+1];
      out[i*3+2]=in[i*3];
    }
  }

}

